#include "hip/hip_runtime.h"
﻿/************************************************************************************************
*						This code is written by Mohammed Ali Alawi Shehab
*						Publication name :Speed Up Needleman-Wunsch Global Alignment Algorithm Using GPU Technique
*						URL				: https://www.researchgate.net/publication/292977570_Speed_Up_Needleman-Wunsch_Global_Alignment_Algorithm_Using_GPU_Technique#feedback/198672
*						Authors			:  Maged Fakirah,  Mohammed A. Shehab,  Yaser Jararweh and Mahmoud Al-Ayyoub
*						INSTITUTION		: Jordan University of Science and Technology, Irbid, Jordan
*						DEPARTMENT		: Department of Computer Science
*************************************************************************************************/
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <string.h>
#include <vector>
#include <iostream>
#include <random>
#include <chrono>
#include <cmath>
#define max(x,y) ((x) > (y) ? (x) : (y))
#define min(x,y)  ((x) < (y) ? (x) : (y))
#define alphabet "ACGT"

// Common Methods
char get_char()
{
    std::random_device rd;
    std::mt19937 gen(rd());

    // Define a uniform distribution in the range [0, RAND_MAX]
    std::uniform_int_distribution<int> distribution(0, 3);
    int rand_index = distribution(gen);
    return alphabet[rand_index];
}

const char* generate_sequence(int size)
{
    std::string result = "";
    for (int i = 0; i < size; i++)
    {
        result.push_back(get_char());
    }

    return result.c_str();
}

// CPU AD Methods
int get_original_row(int num_of_cols, int ad_row_index, int ad_cell_index)
{
    return ad_row_index >= num_of_cols ? ad_row_index - num_of_cols + ad_cell_index + 1 : ad_cell_index;
}

int get_original_column(int num_of_cols, int ad_row_index, int ad_cell_index)
{
    return min(ad_row_index, num_of_cols - 1) - ad_cell_index;
}

int get_cell_score(char x, char y, int score)
{
    return x == y ? score : -score;
}

std::vector<std::vector<int>> split_into_anti_diagonal_rows(const std::vector<std::vector<int>>& matrix) {
    int m = matrix.size();
    int n = matrix[0].size();
    std::vector<std::vector<int>> anti_diagonals;

    // Create a flipped matrix
    std::vector<std::vector<int>> flipped_matrix(m, std::vector<int>(n, 0));
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            flipped_matrix[i][j] = matrix[i][n - 1 - j];
        }
    }

    for (int d = 0; d < m + n - 1; ++d) {
        int offset = n - 1 - d;
        std::vector<int> anti_diagonal;

        for (int i = 0; i < m; ++i) {
            int j = i + offset;
            if (j >= 0 && j < n) {
                anti_diagonal.push_back(flipped_matrix[i][j]);
            }
        }

        anti_diagonals.push_back(anti_diagonal);
    }

    return anti_diagonals;
}

std::vector<std::vector<int>> sequence_alignment_cpu(std::string sequence_1, std::string sequence_2)
{
    int gap_penalty = -2;
    int score = 1;

    int m = sequence_1.length();
    int n = sequence_2.length();

    std::vector<std::vector<int>> score_matrix(m + 1, std::vector<int>(n + 1, 0));

    std::vector<std::vector<int>> ad_rows = split_into_anti_diagonal_rows(score_matrix);

    for (int i = 0; i < ad_rows.size(); ++i) {
        // In every iteration, initialize two other anti-diagonals needed for calculation of the current anti-diagonal
        std::vector<int>& row_d = (i > 1) ? ad_rows[i - 2] : std::vector<int>(m + 1, 0);
        std::vector<int>& row_hv = (i > 1) ? ad_rows[i - 1] : std::vector<int>(m + 1, 0);
        std::vector<int>& row_current = ad_rows[i];

        // Iterate through elements of the current ad
        for (int j = 0; j < row_current.size(); ++j) {
            // To calculate the current cell's score, obtain the original position of that element inside the matrix
            int original_i = get_original_row(n + 1, i, j);
            int original_j = get_original_column(n + 1, i, j);

            // Former
            // c[j] = hv[j-1], hv[j], d[j-1]
            if (i < n + 1) {
                if (original_i == 0 || original_j == 0) {
                    row_current[j] = i * gap_penalty;
                }
                else {
                    int cell_score = get_cell_score(sequence_1[original_i - 1], sequence_2[original_j - 1], score);
                    row_current[j] = max(row_d[j - 1] + cell_score, max(row_hv[j - 1] + gap_penalty, row_hv[j] + gap_penalty));
                }
            }
            // Mid
            // c[j] = hv[j], hv[j+1], d[j]
            else if (i == n + 1) {
                if (original_i == 0 || original_j == 0) {
                    row_current[j] = i * gap_penalty;
                }
                else {
                    int cell_score = get_cell_score(sequence_1[original_i - 1], sequence_2[original_j - 1], score);
                    row_current[j] = max(row_d[j] + cell_score, max(row_hv[j] + gap_penalty, row_hv[j + 1] + gap_penalty));
                }
            }
            // Latter
            // c[j] = hv[j], hv[j+1], d[j+1]
            else {
                if (original_i == 0 || original_j == 0) {
                    row_current[j] = i * gap_penalty;
                }
                else {
                    int cell_score = get_cell_score(sequence_1[original_i - 1], sequence_2[original_j - 1], score);
                    row_current[j] = max(row_d[j + 1] + cell_score, max(row_hv[j] + gap_penalty, row_hv[j + 1] + gap_penalty));
                }
            }
        }
    }

    return ad_rows;
}

// GPU AD Methods
__device__ int device_min(int x, int y)
{
    return x < y ? x : y;
}

__device__ int device_max(int x, int y)
{
    return x > y ? x : y;
}

__device__ int device_get_original_row(int num_of_cols, int ad_row_index, int ad_cell_index)
{
    return ad_row_index >= num_of_cols ? ad_row_index - num_of_cols + ad_cell_index + 1 : ad_cell_index;
}

__device__ int device_get_original_column(int num_of_cols, int ad_row_index, int ad_cell_index)
{
    return device_min(ad_row_index, num_of_cols - 1) - ad_cell_index;
}

__device__ int device_get_cell_score(char x, char y, int score)
{
    return x == y ? score : -score;
}

__global__ void ad_kernel(char* subsequence_1, char* subsequence_2, int* row_current, int* row_d, int* row_hv, int current_ad_size, int current_row_index, int m, int n, int score, int gap_penalty)
{
    int i = current_row_index;
    int j = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (j >= current_ad_size)
    {
        return;
    }

    int original_i = device_get_original_row(n + 1, i, j);
    int original_j = device_get_original_column(n + 1, i, j);

    // Former
    // c[j] = hv[j - 1], hv[j], d[j - 1]
    if (i < n + 1)
    {
        if (original_i == 0 || original_j == 0)
        {
            row_current[j] = i * gap_penalty;
        }
        else
        {
            int cell_score = device_get_cell_score(subsequence_1[original_i - 1], subsequence_2[original_j - 1], score);
            row_current[j] = device_max(row_d[j - 1] + cell_score, device_max(row_hv[j - 1] + gap_penalty, row_hv[j] + gap_penalty));
        }
    }
    // Mid
    // c[j] = hv[j], hv[j+1], d[j]
    else if (i == n + 1)
    {
        if (original_i == 0 || original_j == 0)
        {
            row_current[j] = i * gap_penalty;
        }
        else
        {
            int cell_score = device_get_cell_score(subsequence_1[original_i - 1], subsequence_2[original_j - 1], score);
            row_current[j] = device_max(row_d[j] + cell_score, device_max(row_hv[j] + gap_penalty, row_hv[j + 1] + gap_penalty));
        }
    }
    // Latter
    else
    {
        if (original_i == 0 || original_j == 0)
        {
            row_current[j] = i * gap_penalty;
        }
        else
        {
            int cell_score = device_get_cell_score(subsequence_1[original_i - 1], subsequence_2[original_j - 1], score);
            row_current[j] = device_max(row_d[j + 1] + cell_score, device_max(row_hv[j] + gap_penalty, row_hv[j + 1] + gap_penalty));
        }
    }
}

void initialize_d_hv_rows(int* &row_d_device, int* &row_hv_device)
{
    int* row_d_host = (int*)malloc(sizeof(int));
    row_d_host[0] = 0;

    int* row_hv_host = (int*)malloc(2 * sizeof(int));
    row_hv_host[0] = -2;
    row_hv_host[1] = -2;

    hipMemcpy(row_d_device, row_d_host, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_hv_device, row_hv_host, 2 * sizeof(int), hipMemcpyHostToDevice);
}

int* sequence_alignment_gpu(char* sequence_1, char* sequence_2, int m, int n)
{
    int gap_penalty = -2;
    int score = 1;

    char* sequence_1_device, * sequence_2_device;

    // Allocate memory and initialize subsequences needed for current ad
    hipMalloc(&sequence_1_device, m * sizeof(char));
    hipMalloc(&sequence_2_device, n * sizeof(char));

    hipMemcpy(sequence_1_device, sequence_1, m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(sequence_2_device, sequence_2, n * sizeof(char), hipMemcpyHostToDevice);

    int num_of_ad = (m + 1) + (n + 1) - 1;
    int longest_ad_size = ceil(sqrt(pow(m + 1, 2) + pow(n + 1, 2)));

    int *row_d_device, *row_hv_device, *row_current_device;

    int* row_current_host = (int*)malloc(longest_ad_size * sizeof(int));

    hipMalloc(&row_d_device, longest_ad_size * sizeof(int));
    hipMalloc(&row_hv_device, longest_ad_size * sizeof(int));
    hipMalloc(&row_current_device, longest_ad_size * sizeof(int));

    initialize_d_hv_rows(row_d_device, row_hv_device);

    for (int i = 2; i < num_of_ad; i++)
    {
        int curr_ad_size = i <= (n + 1) ? (min(i, m + 1, n + 1 - i + 1)) + 1 : (min(m + 1, m + 1 + n + 1 - i + 1, i - n - 1)) + 1;
        dim3 grid_size(1);
        dim3 block_size(curr_ad_size);

        ad_kernel << <grid_size, block_size >> > (sequence_1_device, sequence_2_device, row_current_device, row_d_device, row_hv_device, curr_ad_size, i, m, n, score, gap_penalty);
        
        hipMemcpy(row_current_host, row_current_device, curr_ad_size * sizeof(int), hipMemcpyDeviceToHost);
       
        int* old_row_d_device = row_d_device;
        row_d_device = row_hv_device;
        row_hv_device = row_current_device;
        row_current_device = old_row_d_device;
    }

    //hipFree(sequence_1_device);
    //hipFree(sequence_2_device);
    //hipFree(row_d_device);
    //hipFree(row_hv_device);
    //hipFree(row_current_device);

    return row_current_host;
}

int main(int argc, char* argv[])
{
    char* sequence_1 = const_cast<char*>(generate_sequence(200000));
    char* sequence_2 = const_cast<char*>(generate_sequence(200000));

    //std::cout << "Sequence 1: " << sequence_1 << std::endl;
    //std::cout << "Sequence 2: " << sequence_2 << std::endl;

    // CPU Method
    //auto start_cpu = std::chrono::high_resolution_clock::now();

    //std::vector<std::vector<int>> ad_rows = sequence_alignment_cpu(sequence_1, sequence_2);

    //for (int i = 0; i < ad_rows.size(); i++)
    //{
    //    for (int j = 0; j < ad_rows[i].size(); j++)
    //    {
    //        std::cout << ad_rows[i][j] << " ";
    //    }
    //    
    //    std::cout << std::endl;
    //}

    //auto finish_cpu = std::chrono::high_resolution_clock::now();

    //auto microseconds_cpu = std::chrono::duration_cast<std::chrono::microseconds>(finish_cpu - start_cpu);

    //std::cout << "Time in ms (CPU): " << microseconds_cpu.count() << std::endl;

    // GPU Method
    auto start_gpu = std::chrono::high_resolution_clock::now();

    int* result_gpu = sequence_alignment_gpu(sequence_1, sequence_2, 200000, 200000);

    //std::cout << "Alignment score: " << result_gpu[0] << std::endl;

    auto finish_gpu = std::chrono::high_resolution_clock::now();

    auto microseconds_gpu = std::chrono::duration_cast<std::chrono::microseconds>(finish_gpu - start_gpu);

    std::cout << "Time in ms (GPU): " << microseconds_gpu.count() << std::endl;

    //std::cout << "Ratio: " << microseconds_cpu.count() / microseconds_gpu.count();

    //std::cout << "Score matrix (anti-diagonal order): " << std::endl;
    //for (int i = 0; i < ad_rows.size(); i++)
    //{
    //    for (int j = 0; j < ad_rows[i].size(); j++)
    //    {
    //        std::cout << ad_rows[i][j] << " ";
    //    }

    //    std::cout << std::endl;
    //}

}
